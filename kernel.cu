#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>

#define CEIL_DIV(x, y) ((x + y - 1) / y)

#define DEBUG 0
#define BM 64
#define BK 8
#define BN 64
#define TM 8
#define BLOCKDIM_X (BM * BK)

__global__ void sgemm_multi_res(int M, int N, int K, float alpha, const float* A,
    const float* B, float beta, float* C) {
    // compute position in C that this thread is responsible for
    const int threadRow = threadIdx.x / BN;
    const int threadCol = threadIdx.x % BN;
    const int bx = blockIdx.x;
    const int by = blockIdx.y;
    const int x = blockIdx.x * BN + threadRow * TM;
    const int y = blockIdx.y * BM + threadCol;

    __shared__ float As[BM * BK];
    __shared__ float Bs[BK * BN];
    float threadResults[TM] = { 0.0 };

    const int innerRowA = threadIdx.x / BK, innerColA = threadIdx.x % BK;
    const int innerRowB = threadIdx.x / BN, innerColB = threadIdx.x % BN;

    // `if` condition is necessary for when M or N aren't multiples of 32.
    if (x < M && y < N) {
        // advance pointers to the starting positions
        A += bx * BM * K;                   
        B += by * BN;                      
        C += bx * BM * N + by * BN; 

        // the outer loop advances A along the columns and B along
        // the rows until we have fully calculated the result in C.
        for (int bkIdx = 0; bkIdx < K; bkIdx += BK) {
            // Have each thread load one of the elements in A & B from
            // global memory into shared memory.
            As[innerRowA * BK + innerColA] = A[innerRowA * K + innerColA];
            Bs[innerRowB * BN + innerColB] = B[innerRowB * N + innerColB];

            // block threads in this block until cache is fully populated
            __syncthreads();

            // advance pointers onto next chunk
            A += BK;
            B += BK * N;

            // execute the dotproduct on the currently cached block
            for (int dotIdx = 0; dotIdx < BK; ++dotIdx) {
				float Btmp = Bs[dotIdx * BN + threadCol];
                for (int resIdx = 0; resIdx < TM; ++resIdx) {
                    threadResults[resIdx] += As[(threadRow * TM + resIdx) * BK + dotIdx] * Btmp;
                }
            }
            // need to sync again at the end, to avoid faster threads
            // fetching the next block into the cache before slower threads are done
            __syncthreads();
        }

        for (int resIdx = 0; resIdx < TM; ++resIdx) {
			C[(threadRow * TM + resIdx) * N + threadCol] =
				alpha * threadResults[resIdx] + beta * C[(threadRow * TM + resIdx) * N + threadCol];
        }
    }
}

int main() {
#if DEBUG
    size_t N = 1024;
#else
    size_t N = 4092;
#endif
    size_t size = N * N * sizeof(float);
    float* h_A, * h_B, * h_C;
    float* d_A, * d_B, * d_C;

    h_A = (float*)malloc(size);
    h_B = (float*)malloc(size);
    h_C = (float*)malloc(size);

    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);
    hipMalloc(&d_C, size);

    for (int i = 0; i < N * N; ++i) {
        h_A[i] = rand() / (float)RAND_MAX;
        h_B[i] = rand() / (float)RAND_MAX;
    }

    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    dim3 dimGrid(CEIL_DIV(N, BM), CEIL_DIV(N, BN));
    dim3 dimBlock(BLOCKDIM_X);

    sgemm_multi_res << <dimGrid, dimBlock >> > (N, N, N, 1, d_A, d_B, 0, d_C);

    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

#if DEBUG
    // Validate the result
    bool validationFailed = false;
    for (int i = 0; i < N; ++i) {
        for (int j = 0; j < N; j++) {
			float expected = 0;
			for (int k = 0; k < N; ++k) {
				expected += h_A[i * N + k] * h_B[k * N + j];
			}

			if (std::fabs(expected - h_C[i * N + j]) > 1e-3) {
				printf("Validation failed at index %d: expected %f, got %f\n", i * N + j, expected, h_C[i * N + j]);
				validationFailed = true;
			}
        }
    }

    if (!validationFailed) {
        printf("Validation passed!\n");
    }
    else {
        printf("Validation failed!\n");
    }
#endif
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    free(h_A);
    free(h_B);
    free(h_C);

    return 0;
}
