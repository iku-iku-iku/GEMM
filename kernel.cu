#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

__global__ void matrixMulKernel(float* A, float* B, float* C, int N) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    if (row < N && col < N) {
        float sum = 0;
        for (int i = 0; i < N; ++i) {
            sum += A[row * N + i] * B[i * N + col];
        }
        C[row * N + col] = sum;
    }
}

int main() {
    int N = 1024;
    size_t size = N * N * sizeof(float);
    float* h_A, * h_B, * h_C;
    float* d_A, * d_B, * d_C;

    h_A = (float*)malloc(size);
    h_B = (float*)malloc(size);
    h_C = (float*)malloc(size);

    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);
    hipMalloc(&d_C, size);

    // Initialize host matrices
    for (int i = 0; i < N * N; ++i) {
        h_A[i] = rand() / (float)RAND_MAX;
        h_B[i] = rand() / (float)RAND_MAX;
    }

    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid(N / threadsPerBlock.x, N / threadsPerBlock.y);

    matrixMulKernel << <blocksPerGrid, threadsPerBlock >> > (d_A, d_B, d_C, N);

    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    // Validate the result
    // ... (validation code here)

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    free(h_A);
    free(h_B);
    free(h_C);

    return 0;
}
